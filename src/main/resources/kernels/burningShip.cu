
#include <hip/hip_runtime.h>
extern "C"
__global__ void burningShip(double zoom, double posX, double posY, int maxIter, int width, int height, int* argb)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = y * width + x;

    if(x >= width || y >= height){
        return;
    }

    double zPrevRe = 0;
    double zPrevIm = 0;
    double zNextRe = 0;
    double zNextIm = 0;

    double pRe = (x - width/2.0)*zoom+posX;
    double pIm = (y - height/2.0)*zoom+posY;

    int i = 0;
    while(i++ < maxIter-1)
    {
        // zNext = zPrev*zPrev + p
        zNextRe = zPrevRe * zPrevRe - zPrevIm * zPrevIm + pRe;
        zNextIm = 2.0*zPrevRe*zPrevIm + pIm;

        // |zPrev| > 4.0
        if((zNextRe * zNextRe + zNextIm * zNextIm) > 4.0){
            break;
        }

        zPrevRe = fabs(zNextRe);
        zPrevIm = fabs(zNextIm);
    }

    int color = (255.0*i)/maxIter;
    argb[idx] = (255<<24) | (color<<16) | (color<<8) | color;
}
