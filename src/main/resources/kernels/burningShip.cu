
#include <hip/hip_runtime.h>
extern "C"
__global__ void burningShip(double zoom, double posX, double posY, int maxIter, int width, int height, int* argb)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = y * width + x;

    if(x >= width || y >= height){
        return;
    }

    double zPrevRe = 0;
    double zPrevIm = 0;
    double zNextRe = 0;
    double zNextIm = 0;

    double pRe = (x - width/2.0)*zoom+posX;
    double pIm = (y - height/2.0)*zoom+posY;

    int i = 0;
    while(i++ < maxIter-1)
    {
        // zNext = zPrev*zPrev + p
        zNextRe = zPrevRe * zPrevRe - zPrevIm * zPrevIm + pRe;
        zNextIm = 2.0*zPrevRe*zPrevIm + pIm;

        // |zPrev| > 4.0
        if((zNextRe * zNextRe + zNextIm * zNextIm) > 4.0){
            break;
        }

        zPrevRe = fabs(zNextRe);
        zPrevIm = fabs(zNextIm);
    }

    double color = (255.0*i)/(1.0*maxIter);
    int r = 17.0*(abs(255-color)/255.0);
    int g = 255.0*(color/255.0);
    int b = 33.0*(abs(255-color)/255.0);
    argb[idx] = (255<<24) | (r<<16) | (g<<8) | b;
}
