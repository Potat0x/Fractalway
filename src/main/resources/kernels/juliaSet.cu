
#include <hip/hip_runtime.h>
extern "C"
__global__ void juliaSet(double zoom, double posX, double posY, int maxIter, int width, int height, int* argb, double cRe, double cIm)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = y * width + x;

    if(x >= width || y >= height){
        return;
    }

    double zNextRe = 0;
    double zNextIm = 0;

    double pRe = (x - width/2.0)*zoom+posX;
    double pIm = (y - height/2.0)*zoom+posY;

    double zPrevRe = pRe;
    double zPrevIm = pIm;

    int i = 0;
    while(i++ < maxIter-1)
    {
        // zNext = zPrev*zPrev + c
        zNextRe = zPrevRe * zPrevRe - zPrevIm * zPrevIm + cRe;
        zNextIm = 2.0*zPrevRe*zPrevIm + cIm;

        // |zPrev| > 4.0
        if((zNextRe * zNextRe + zNextIm * zNextIm) > 4.0){
            break;
        }

        zPrevRe = zNextRe;
        zPrevIm = zNextIm;
    }

    double color = (255.0*i)/(1.0*maxIter);
    int r = 17.0*(abs(255-color)/255.0);
    int g = 255.0*(color/255.0);
    int b = 33.0*(abs(255-color)/255.0);
    argb[idx] = (255<<24) | (r<<16) | (g<<8) | b;
}
